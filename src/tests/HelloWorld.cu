#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include "tests_common.cuh"
#include "cudawrapper.h"

int main(int argc, char* argv[])
{
  int size = 32*1024*1024;

  int localRank, globalRank, numRanks;

  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;
  CudaWrapper* cudaWrapper_ = ncclSetupWrappers(false);

  std::tie(localRank, globalRank, numRanks, comm) = setupNccl(argc, argv);

  CUDACHECK_TEST(
      cudaWrapper->hipMalloc((void**)&sendbuff, size * sizeof(float)));
  CUDACHECK_TEST(
      cudaWrapper->hipMalloc((void**)&recvbuff, size * sizeof(float)));
  CUDACHECK_TEST(cudaWrapper->hipStreamCreate(&s));

  //communicating using NCCL
  NCCLCHECK_TEST(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum,
        comm, s));

  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK_TEST(cudaWrapper->hipStreamSynchronize(s));

  //free device buffers
  CUDACHECK_TEST(cudaWrapper->hipFree(sendbuff));
  CUDACHECK_TEST(cudaWrapper->hipFree(recvbuff));

  cleanupNccl(comm);

  printf("[MPI Rank %d] Success \n", globalRank);
  return 0;
}
